#include <stdio.h>
#include <iostream>
#include <ctime>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_BLOCKS 16
#define NUM_THREADS 16
#define Num_Queens 8
#define MAX_ITER 4000

using namespace std;

__device__ int checkDiagonals(int q,int i, int* S)
// Returns 1 if no queen in diagonal, else 0
{
	int I = blockIdx.x*NUM_THREADS*Num_Queens + threadIdx.x*Num_Queens;
	for (int j = 1; j<=i; j++){
		if (S[I+i-j] == q-j | S[I+i-j] == q+j){
			return 0;
		}
	}
	return 1;
}

__device__ int sum(int row[], int len)
// Returns sum of an array
{

	int s = 0;
	for (int i = 0; i<len; i++){
		s += row[i];
	}
	return s;
}

/*__global__ void setup_kernel (curandState * state, unsigned long seed)
// Create states to generate random numbers
{
	int id = blockIdx.x*NUM_BLOCKS + threadIdx.x;
	curand_init( seed, id, 0, &state[id] );
}
*/
__global__ void kernel(int* Sol, hiprandState* globalState,unsigned long seed)
// Kernel to solve puzzle
{
	int ind = blockIdx.x*NUM_BLOCKS + threadIdx.x;
	hiprand_init( seed, ind, 0, &globalState[ind] );

	// Index for thread to store solution
	int I = blockIdx.x*NUM_THREADS*Num_Queens + threadIdx.x*Num_Queens;
	//int ind = blockIdx.x*NUM_BLOCKS + threadIdx.x;
	int d_Placement[Num_Queens];				// Rows where queens is placed. 1 = row taken
	int tried[Num_Queens][Num_Queens];				// Positions tried at column i




	int queen;

	// Initialize variables
	for (int i = 0; i < Num_Queens; i++){
		Sol[I+i] = -1;
		 d_Placement[i] = 0;
		for (int j = 0; j < Num_Queens; j++){
			tried[i][j] = 0;
		}
	}

	// Set start column and iter counter
  int	i = 0;
	int iter = 0;

	// Get local state to generate numbers
	hiprandState localState = globalState[ind];

	while (iter < Num_Queens)
	{

		// Generate random number
		queen = hiprand_uniform( &localState ) * Num_Queens;

		if ( d_Placement[queen] == 0 & tried[i][queen] == 0){ 		// Row clear and not tried before
			tried[i][queen] = 1;				// Set position as tried
			if (checkDiagonals(queen,i,Sol)==1){	// If no attacking queens in diagonal
				Sol[I+i] = queen;			// Add queen to solution
			  d_Placement[queen] = 1;			// Set row as taken
				i++;				// Increment interation counter
				if (i == Num_Queens){			// Finished!
					break;
				}
			}
		}
		if (sum(tried[i],Num_Queens) + sum( d_Placement,Num_Queens) == Num_Queens){ 		// All positions tried
			 d_Placement[Sol[I+i-1]] = 0;					// Free domain
			Sol[I+i-1] = -1;						// Remove queen from solution

			for (int j = 0; j<Num_Queens; j++){		// Reset positions tried for column
				tried[i][j] = 0;
			}
			i--;				// Backtrack to prevoius column
		}
		iter++;
	}
}

int main()
{
	// Initialize states variable and allocate memory
	hiprandState* devStates;
	hipMalloc ( &devStates, Num_Queens*sizeof( hiprandState ) );

	// Initialze seeds
	//setup_kernel <<< NUM_BLOCKS, NUM_THREADS>>> ( devStates,unsigned(time(NULL)) );
	//int id = blockIdx.x*NUM_BLOCKS + threadIdx.x;
	//curand_init( unsigned(time(NULL)), id, 0, &devStates[id] );

	// Initialize array to store solution
	int solution_host[Num_Queens*NUM_BLOCKS*NUM_THREADS];
	int* solution_dev;

	// Allocate memory on device
	hipMalloc((void**) &solution_dev, (sizeof(Num_Queens*NUM_BLOCKS*NUM_THREADS)));

	// Start clock
	clock_t begin = clock();
	// Launch kernel on device
	kernel<<<NUM_BLOCKS,NUM_THREADS>>> (solution_dev, devStates, unsigned(time(NULL)));
	// Copy solution from device to host
	hipMemcpy(solution_host, solution_dev, sizeof(int)*Num_Queens*NUM_BLOCKS*NUM_THREADS, hipMemcpyDeviceToHost);
	// End clock
	clock_t end = clock();

	double elapsed_sec = double(end - begin)/(CLOCKS_PER_SEC/1000);

	// Print time used
	cout << elapsed_sec << endl;

	// Count solutions found (not -1 in last position)
	int solution_count = 0;
	for(int l = 0; l <= (sizeof(solution_host) / sizeof(int)); l++){
		if (solution_host[l] != -1){
			solution_count++;
		}
	}
	// Print solutions found
	cout << solution_count << endl;

	// Free memory on device
	hipFree(devStates);
	hipFree(solution_dev);

	return 0;
}
