/* ==================================================================
  Programmers: Conner Wulf (connerwulf@mail.usf.edu),
               Derek Rodriguez (derek23@mail.usf.edu)
	       David Hoambrecker (david106@mail.usf.edu)
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <vector>

using namespace std;
static int total = 0;
struct timezone Idunno;	
struct timeval startTime, endTime;
//CPU helper function to test is a queen can be placed
int isAllowed(vector<vector<int> > board, int row, int col, int n)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}

// GPU helper problem
/*
__device__ int isAllowedGpu(, int row, int col)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}
*/

//N-queen solver for CPU algorithm
int Solver(vector<vector<int> > board, int col, int n)
{
  if (col >= n)
  {

    total++;
    return 1;
  }

  int nextState = 0;
  for(int k = 0; k < n; k++)
  {
    if(isAllowed(board,k,col,n))
    {
      board[k][col] = 1;
      nextState = Solver(board, col + 1,n) || nextState;
      board[k][col] = 0;
    }
  }

  return nextState;
}

double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

int main(int argc, char **argv) {
  //  CPU VERSION
  
	const int n = atoi(argv[1]);
	vector<vector<int> > board;
	board.resize(n, std::vector<int>(n, 0));
	
	srand(1);
	gettimeofday(&startTime, &Idunno);

  if(Solver(board,0,n) == 0)
  {
    printf("No Solution\n");
  	report_running_time();
    return 0;
  }
  printf("\nTotal Solutions(CPU): %d boards\n\n",total);
	report_running_time();

  return 0;

}
