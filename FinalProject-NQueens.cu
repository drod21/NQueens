/* ==================================================================
  Programmers: Conner Wulf (connerwulf@mail.usf.edu),
               Derek Rodriguez (derek23@mail.usf.edu)
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime.h>


const int n = 6;

void outputSolution(int board[n][n]) {
    static int k = 1;

    printf("%d-\n",k++);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf(" %d ", board[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}

//CPU helper function to test is a queen can be placed
int isAllowed(int board[n][n], int row, int col)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}

// GPU helper problem
__device__ int isAllowed(int board[n][n], int row, int col)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}

int solverUtil(int board[n][n], int col)
{
  int count = 0;
  int nextState = 0;

  for(int k = 0; k < n; k++)
  {
    for(int j = 0; j < n; j++) {
      if (col == n)
      {
        count++;
        outputSolution(board);
        printf("count: %d\n", count);
        nextState = 1;
      }

      if (isAllowed(board, k, col))
      {
        board[k][col] = 1;
      }
    }
  }
  return nextState;
}

//N-queen solver for CPU algorithm
int Solver(int board[n][n], int col)
{
  int count = 0;
  if (col == n)
  {
    count++;
      outputSolution(board);
      printf("count: %d\n", count);
    return 1;
  }

  int nextState = 0;

  for(int k = 0; k < n; k++)
  {
    if (isAllowed(board,k,col))
    {
      board[k][col] = 1;
      nextState = Solver(board, col + 1);
      board[k][col] = 0;
    }
  }
  return nextState;
}

int main(int argc, char **argv) {

  //  n = atoi(argv[1]);
  int board[n][n];
  memset(board,0,sizeof(board));

  if(Solver(board,0) == 0)
  {
    printf("No Solution\n");
    return 0;
  }

  return 0;
}
