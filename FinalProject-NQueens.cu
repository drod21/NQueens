/* ==================================================================
	Programmers: Conner Wulf (connerwulf@mail.usf.edu)
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime.h>


const int n = 4;
//CPU helper function to test is a queen can be placed
int isAllowed(int board[n][n], int row, int col)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x])
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y])
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y])
    {
      return 0;
    }
  }
 return 1;
}

//N-queen solver for CPU algorithm
int Solver(int board[n][n], int col)
{
  int count = 0;
  if (col == n)
  {
    count++;
    return 1;
  }

  int nextState = 0;

  for(int k = 0; k < n; k++)
  {
    if (isAllowed(board,k,col))
    {
      board[k][col] = 1;
      nextState = Solver(board, col + 1);
      board[k][col] = 0;
    }
  }
  return nextState;
}

int main(int argc, char **argv){

//  n = atoi(argv[1]);
  int board[n][n];
  memset(board,0,sizeof(board));

  if(Solver(board,0) == 0)
  {
    printf("No Solution");
    return 0;
  }

  return 0;
}
