/* ==================================================================
  Programmers: Conner Wulf (connerwulf@mail.usf.edu),
               Derek Rodriguez (derek23@mail.usf.edu)
	       David Hoambrecker (david106@mail.usf.edu)
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <vector>

using namespace std;
static int total = 0;
unsigned long count = 0;
struct timezone Idunno;	
struct timeval startTime, endTime;
//CPU helper function to test is a queen can be placed
int isAllowed(int **board, int row, int col, int n) // make this the kernel?????
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}

// GPU helper problem

/*
__global__ void nqueen_kernel_3(*job_data, *results ... )
{
  __const__ tid; //The index of the thread within the block
  __register__ rowIndex, solution, index;
  __shared__ ROW[MAX_ROW][BLOCK_SIZE];
  __const__ upper_bound = the upper bound of the job-pool for this block;
  __shared__ seek;
  if(tid == 0) { set seek to point to the next new job in the job-pool for this block;}
  each thread fetches a task from job_data into its array ROW[MAX_ROW] [tid];
  for(; rowIndex >= 0; rowIndex--) {
  ĂĂ //the same code as in the low-divergence n-queens kernel is omitted
  if (rowIndex == 0) { //current job is done.
  index = atomicAdd(&seek,1); //get index of new job
  if (index exceeds pool upper bound)
  break;
  else{
  gets this job by index from the job-pool as the new job of this thread;
  rowIndex++;
  }
  }
  }
  reduction of the solutions of the threads within this block;
}
*/

/* use this one */
// /*
// __global__ void nqueen_kernel_0(int *job_data, int *results, int *work_space)
// {
//   __register__ rowIndex, solution;
//   int tx = threadIdx.x;
//   int x = tx * blockDim.x + threadIdx.x;

//   each thread fetches a task from job_data into its array ROW[ ] in work_space;
//   while(rowIndex >= 0) {
//     if (no position to place new queen in ROW[rowIndex]) { rowIndex--; }
//     else{
//       finds a valid position P in ROW[rowIndex];
//       places a queen at P in ROW[rowIndex] and mark the position as occupied;
//     if (reaches last row) { solution++; }
//     else{
//       generates ROW[rowIndex+1] based on Row[rowIndex] and the position P;
//       rowIndex++;
//     }
//   }
// }
// reduction of the solutions of the threads within each bl*/


// __global__ void queenSolverGpu(int *d_board, int n, int *allowed, int *count) {
//   int threadId = blockIdx.x * blockDim.x + threadIdx.x;
//   int threadX = threadIdx.x;
//   int qBitCol[n * blockDim.x];
//   int qBitPosDiag[n * blockDim.x];
//   int qBitNegDiag[n * blockDim.x];
//   int stack[n*n+2];
//   register int nStack;
//   qBitCol[tx]=qBitPosDiag[tx]=qBitNegDiag[tx]=0;
// }
//N-queen solver for CPU algorithm
// int SolverGPU(int **board, int col, int n)
// {
//   int *allowed;
//   int temp = 0;
//   int *d_board;
//   dim3 threadsPerBlock(n, n);
//   dim3 numBlocks(n / threadsPerBlock.x, n / threadsPerBlock.y);
//   cudaMalloc((void **) &d_board, sizeof(int) * n);
//   cudaMalloc((void **) &allowed, n);
//   cudaMemcpy(allowed, &temp, sizeof(int), cudaMemcpyHostToDevice);
//   cudaMemcpy(d_board, board[0], sizeof(int) * n * n, cudaMemcpyHostToDevice);

//   if (col >= n * n)
//   {
//     total++;
//     return 1;
//   }

//   int nextState = 0;
//   for(int k = 0; k < n; k++)
//   {
//     isAllowedGpu<<<numBlocks, threadsPerBlock>>>(d_board, k, col, n * n, allowed);
//     cudaMemcpy(&temp, allowed, sizeof(int), cudaMemcpyDeviceToHost);
//     if(temp == 1)
//     {
//       board[k][col] = 1;
//       nextState = Solver(board, col + 1, n) || nextState;
//       board[k][col] = 0;
//     }
//   }

//   return nextState;
// }

int Solver(int **board, int col, int n)
{
  if (col >= n)
  {
    total++;
    return 1;
  }

  int nextState = 0;

  for(int k = 0; k < n; k++)
  {
    if (isAllowed(board,k,col, n))
    {
      board[k][col] = 1;
      nextState = Solver(board, col + 1, n);
      board[k][col] = 0;
    }
  }
  return nextState;
}

double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

int main(int argc, char **argv) {
  //  CPU VERSION
  
  const int n = atoi(argv[1]);
  int **board;
  int **newHostBoard;
  newHostBoard = (int **) malloc(n * sizeof(int *));
  board = (int **) malloc(n * sizeof(int *));
  for (int i = 0; i < n; i++) {
    board[i] = (int *) malloc(n * sizeof(int));
    newHostBoard[i] = (int *) malloc(n * sizeof(int));
  }
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      board[i][j] = 0;
    }
  }
	// vector<vector<int> > board;
  // board.resize(n, std::vector<int>(n, 0));
  // int *allowed;
  // int temp = 0;
  // int *d_board;
  // int *count;
  // size_t pitch;
  // dim3 threadsPerBlock(n, 1, 1);
  // dim3 numBlocks(n / threadsPerBlock.x, 1, 1);
  // cudaMalloc((void **) &allowed, n);
  // cudaMalloc((void **) &count, n);
  // cudaMallocPitch(&d_board, &pitch, n * sizeof(int), n);
  // cudaMemcpy2D(d_board, pitch, board, n * sizeof(int), n * sizeof(int), n, cudaMemcpyHostToDevice);
  // cudaMemcpy(allowed, &temp, sizeof(int), cudaMemcpyHostToDevice);
  // cudaMemcpy(count, &total, sizeof(int), cudaMemcpyHostToDevice);
  // // for(int i = 0; i < n; i++) {
  // //   for(int k = 0; k < n; k++) {
  //     isAllowedGpu<<<numBlocks, threadsPerBlock, n>>>(d_board, n, 0, n, allowed, count);
  //     cudaMemcpy(&temp, allowed, sizeof(int), cudaMemcpyDeviceToHost);
  //     if(temp == 1) {
  //       newHostBoard[i][k] = 1;

  //     }
  //   }
  // }
  // cudaMemcpy(&total, count, sizeof(int), cudaMemcpyDeviceToHost);
  // cudaMemcpy2D(newHostBoard, pitch, d_board, n * sizeof(int), n * sizeof(int), n, cudaMemcpyDeviceToHost);
	
	
  // if(temp == 0) {
  //   printf("No Solution\n");
  //   report_running_time();
  //   return 0;
  // }

	srand(1);
  gettimeofday(&startTime, &Idunno);
  Solver(board, 0, n);
  
  // if(Solver(board,0,n) == 0)
  // {
  //   printf("No Solution\n");
  // 	report_running_time();
  //   return 0;
  // }
  printf("\nTotal Solutions(CPU): %d boards\n\n",total);
  report_running_time();

  return 0;

}
