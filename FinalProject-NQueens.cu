/* ==================================================================
  Programmers: Conner Wulf (connerwulf@mail.usf.edu),
               Derek Rodriguez (derek23@mail.usf.edu)
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime.h>


const int n = 8;
static int total = 0;



//CPU helper function to test is a queen can be placed
int isAllowed(int board[n][n], int row, int col)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}

// GPU helper problem
__device__ int isAllowedGpu(int board[n][n], int row, int col)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}

//N-queen solver for CPU algorithm
int Solver(int board[n][n], int col)
{
  if (col >= n)
  {

    total++;
    return 1;
  }

  int nextState = 0;
  for(int k = 0; k < n; k++)
  {
    if(isAllowed(board,k,col))
    {
      board[k][col] = 1;
      nextState = Solver(board, col + 1) || nextState;
      board[k][col] = 0;
    }
  }

  return nextState;
}

int main(int argc, char **argv) {
  //  CPU VERSION
  int board[n][n];
  memset(board,0,sizeof(board));

  if(Solver(board,0) == 0)
  {
    printf("No Solution\n");
    return 0;
  }
  printf("\nTotal Solutions(CPU): %d boards\n\n",total);
  return 0;

}
