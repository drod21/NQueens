#include "hip/hip_runtime.h"
/* ==================================================================
  Programmers: Conner Wulf (connerwulf@mail.usf.edu),
               Derek Rodriguez (derek23@mail.usf.edu)
	       David Hoambrecker (david106@mail.usf.edu)
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;
static int total = 0;
unsigned long count = 0;
long *answer;
struct timezone Idunno;
struct timeval startTime, endTime;

 #ifndef NUM
 #define NUM 12
 #endif

//CPU helper function to test is a queen can be placed
int isAllowed(int **board, int row, int col, int n)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}
// CPU Solver for N-queens problem
int Solver(int **board, int col, int n)
{
  if (col >= n)
  {
    total++;
    return 1;
  }

  int nextState = 0;

  for(int k = 0; k < n; k++)
  {
    if (isAllowed(board,k,col, n))
    {
      board[k][col] = 1;
      nextState = Solver(board, col + 1, n);
      board[k][col] = 0;
    }
  }
  return nextState;
}

// GPU parallel kernel for N-Queens
__global__ void kernel(long *answer, int SegSize, int nBX, int nBY, int genNum)
{
  __shared__ long sol[NUM][NUM];
  __shared__ char tup[NUM][NUM][NUM];

  int wrongCount = 0;
  sol[threadIdx.x][threadIdx.y] = 0;
  tup[threadIdx.x][threadIdx.y][0] = blockIdx.y % SegSize;
  int totalGenerated = powf(NUM, genNum);
  int blockYSeg = blockIdx.y / SegSize;
  int workLoad = totalGenerated / nBY;
  int runOff = totalGenerated - workLoad *nBY;




  int temp = blockIdx.x;
  for(int x = 1; x <=nBX; x++)
  {
    tup[threadIdx.x][threadIdx.y][x] = temp % NUM;
    temp = temp / NUM;

  }

  int tupCount = nBX;
  tup[threadIdx.x][threadIdx.y][++tupCount] = threadIdx.x;
  tup[threadIdx.x][threadIdx.y][++tupCount] = threadIdx.y;

  for(int k = tupCount; k > 0; k--)
  {
    for(int m = k - 1, counter = 1; m >= 0; counter++, m--)
    {
      //Checks diagonal left, down
      wrongCount += (tup[threadIdx.x][threadIdx.y][k] + counter) == tup[threadIdx.x][threadIdx.y][m];
      //Checks row its in
      wrongCount += tup[threadIdx.x][threadIdx.y][k] == tup[threadIdx.x][threadIdx.y][m];
      // Checks diagonal left, up
      wrongCount  += (tup[threadIdx.x][threadIdx.y][k] - counter) == tup[threadIdx.x][threadIdx.y][m];

    }
  }






  if (wrongCount == 0)
  {
    int begin = blockYSeg * workLoad;
    for(int c = begin; c < begin + workLoad + (blockYSeg == nBY - 1) * runOff; c++)
    {
      //last values is made in tuple, convert and store to tup array
      int temp = c;
      for(int q = 0, z =tupCount + 1; q < genNum; z++, q++)
      {
        tup[threadIdx.x][threadIdx.y][q] = temp % NUM;
        temp = temp / NUM;
      }

      //checks that the genNum tuple values are indeed unique (saves work overall)
      for(int a = 0; a < genNum && wrongCount == 0; a++){
				for(int b = 0; b < genNum && wrongCount == 0; b++){
					wrongCount += tup[threadIdx.x][threadIdx.y][tupCount + 1 + a] == tup[threadIdx.x][threadIdx.y][tupCount + 1 + b] && a != b;
				}
			}

      for(int k = NUM -1; k > wrongCount; k--)
      {
        for(int m = k - 1, counter = 1; m >= 0; counter++, m--)
        {
          //Checks diagonal left, down
          wrongCount += (tup[threadIdx.x][threadIdx.y][k] + counter) == tup[threadIdx.x][threadIdx.y][m];
          //Checks row its in
          wrongCount += tup[threadIdx.x][threadIdx.y][k] == tup[threadIdx.x][threadIdx.y][m];
          // Checks diagonal left, up
          wrongCount  += (tup[threadIdx.x][threadIdx.y][k] - counter) == tup[threadIdx.x][threadIdx.y][m];

        }
      }

      sol[threadIdx.x][threadIdx.y] += !(wrongCount);
      wrongCount = 0;

    }
  }

  __syncthreads();

    // sum all threads in block to get total
  	if(threadIdx.x == 0 && threadIdx.y == 0)
    {

  		long total = 0;

  		for(int i =0; i < NUM; i++){
  			for(int j = 0; j < NUM; j++){
  				total += sol[i][j];
  			}
  		}
  		answer[gridDim.x * blockIdx.y + blockIdx.x] = total;
  	}


  	__syncthreads();
}


double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("CPU Time: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}


int main(int argc, char **argv) {

  if(argc < 3) {

    printf("\nError, too few arguments. Usage: ./CHANGE THIS\n");
    return -1;
  }

  const int NUM_TUPLEX = atoi(argv[1]);
  const int NUM_TUPLEY = atoi(argv[2]);
  const int generatedNum = NUM - 3 - NUM_TUPLEX;
  hipEvent_t start, stop;
  float elapsedTime;

  if(generatedNum < 0){
    printf("\nThe numbers generated iteratively cannot be less than 0.\n");
    exit(1);
  }

  //ensure N is in the correct range
  if(NUM < 4  || NUM > 22){
    printf("\nN(%d) must be between 4 and 22 inclusive\n", NUM);
    exit(1);
  }

  //ensure that at least one of the tuple values is generated by the block's X coordinate value
  if(NUM_TUPLEX < 1){
    printf("\nThe number of tuples generated by each block's X coordinate value must be >= 1\n");
    exit(1);
  }

  	//ensure that the number of Y segments that the numGen work is divided into
  	//is at least one per work segment
  	if(NUM_TUPLEY > pow(NUM, generatedNum)){
  		printf("\n number of groups of columns must be less than or equal to N^(N - 3 - (1st ARG))\n");
  		exit(1);
  	}

  //CPU setup
  int **board;
  board = (int **) malloc(NUM * sizeof(int *));

  for (int i = 0; i < NUM; i++) {
    board[i] = (int *) malloc(NUM * sizeof(int));

  }
  for (int i = 0; i < NUM; i++) {
    for (int j = 0; j < NUM; j++) {
      board[i][j] = 0;

    }
  }

  int WIDTH, HEIGHT, NUM_BLOCKS, YSegmentSize;
  WIDTH = pow(NUM, NUM_TUPLEX);
  YSegmentSize = (NUM / 2) + (NUM % 2);
  HEIGHT = YSegmentSize + NUM_TUPLEY;
  NUM_BLOCKS = WIDTH * HEIGHT;


  long *d_answer;
  answer = new long[NUM_BLOCKS];

  hipMalloc((void **) &d_answer, sizeof(long) * NUM_BLOCKS);

  dim3 block(NUM, NUM); //threads w x h
  dim3 grid(WIDTH, HEIGHT); //blocks w x h

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  kernel<<<grid, block>>>(d_answer, YSegmentSize, NUM_TUPLEX, NUM_TUPLEY, generatedNum);
  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  hipMemcpy(answer,d_answer, sizeof(long) * NUM_BLOCKS, hipMemcpyDeviceToHost);





	srand(1);
  gettimeofday(&startTime, &Idunno);
  Solver(board, 0, NUM);


  printf("\nTotal Solutions: %d boards\n\n",total);
  report_running_time();
  printf("GPU Time: %f secs\n", (elapsedTime / 1000.00));
  return 0;

}
