/* ==================================================================
  Programmers: Conner Wulf (connerwulf@mail.usf.edu),
               Derek Rodriguez (derek23@mail.usf.edu)
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime.h>


const int n = 8;
static int total = 0;
struct timezone Idunno;	
struct timeval startTime, endTime;
//CPU helper function to test is a queen can be placed
int isAllowed(int board[n][n], int row, int col)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}

// GPU helper problem
__device__ int isAllowedGpu(int board[n][n], int row, int col)
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}

//N-queen solver for CPU algorithm
int Solver(int board[n][n], int col)
{
  if (col >= n)
  {

    total++;
    return 1;
  }

  int nextState = 0;
  for(int k = 0; k < n; k++)
  {
    if(isAllowed(board,k,col))
    {
      board[k][col] = 1;
      nextState = Solver(board, col + 1) || nextState;
      board[k][col] = 0;
    }
  }

  return nextState;
}

double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

int main(int argc, char **argv) {
  //  CPU VERSION
  int board[n][n];
  memset(board,0,sizeof(board));
	srand(1);
	gettimeofday(&startTime, &Idunno);

  if(Solver(board,0) == 0)
  {
    printf("No Solution\n");
  	report_running_time();
    return 0;
  }
  printf("\nTotal Solutions(CPU): %d boards\n\n",total);
	report_running_time();

  return 0;

}
