/* ==================================================================
  Programmers: Conner Wulf (connerwulf@mail.usf.edu),
               Derek Rodriguez (derek23@mail.usf.edu)
	       David Hoambrecker (david106@mail.usf.edu)
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <vector>

using namespace std;
static int total = 0;
struct timezone Idunno;	
struct timeval startTime, endTime;
//CPU helper function to test is a queen can be placed
int isAllowed(int **board, int row, int col, int n) // make this the kernel?????
{
  int x,y;

  //left check
  for (x = 0; x < col; x++)
  {
    if(board[row][x] == 1)
    {
      return 0;
    }
  }
  //check left diagonal up
  for(x = row, y = col; x >= 0 && y >= 0; x--, y--)
    {
      if (board[x][y] == 1)
      {
        return 0;
      }
    }
  for(x = row, y = col; x < n && y >= 0; x++, y--)
  {
    if  (board[x][y] == 1)
    {
      return 0;
    }
  }
 return 1;
}

// GPU helper problem

/*
__global__ void nqueen_kernel_3(*job_data, *results ... )
{
  __const__ tid; //The index of the thread within the block
  __register__ rowIndex, solution, index;
  __shared__ ROW[MAX_ROW][BLOCK_SIZE];
  __const__ upper_bound = the upper bound of the job-pool for this block;
  __shared__ seek;
  if(tid == 0) { set seek to point to the next new job in the job-pool for this block;}
  each thread fetches a task from job_data into its array ROW[MAX_ROW] [tid];
  for(; rowIndex >= 0; rowIndex--) {
  ĂĂ //the same code as in the low-divergence n-queens kernel is omitted
  if (rowIndex == 0) { //current job is done.
  index = atomicAdd(&seek,1); //get index of new job
  if (index exceeds pool upper bound)
  break;
  else{
  gets this job by index from the job-pool as the new job of this thread;
  rowIndex++;
  }
  }
  }
  reduction of the solutions of the threads within this block;
}
*/

/* use this one */
/*
__global__ void nqueen_kernel_0(*job_data, *results, *work_space... )
{
  __register__ rowIndex, solution;
  each thread fetches a task from job_data into its array ROW[ ] in work_space;
  while(rowIndex >= 0) {
    if (no position to place new queen in ROW[rowIndex]) { rowIndex--; }
    else{
      finds a valid position P in ROW[rowIndex];
      places a queen at P in ROW[rowIndex] and mark the position as occupied;
    if (reaches last row) { solution++; }
    else{
      generates ROW[rowIndex+1] based on Row[rowIndex] and the position P;
      rowIndex++;
    }
  }
}
reduction of the solutions of the threads within each bl*/
__global__ void isAllowedGpu(int *d_board, int row, int col, int n, int *allowed)
{
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  int i, j;
  //left check
  for (i = x; i < col + x; i++)
  {
    if(d_board[i] == 1)
    {
      *allowed = 0;
      return;
    }
  }
  //check left diagonal up
  for(i = row + y, j = x + col; i >= y && j >= x; i--, j--)
    {
      if(d_board[i * n + j] == 1)
      {
        *allowed = 0;
        return;
      }
    }
  for(i = row, j = col + x; i < n && j >= x; i++, j--)
  {
    if(d_board[i * n + j] == 1)
    {
      *allowed = 0;
      return;
    }
  }
 *allowed = 1;
}
//N-queen solver for CPU algorithm
int Solver(int **board, int col, int n)
{
  int *allowed;
  int temp = 0;
  int *d_board;
  dim3 threadsPerBlock(n, n);
  dim3 numBlocks(n / threadsPerBlock.x, n / threadsPerBlock.y);
  hipMalloc((void **) &d_board, sizeof(int) * n);
  hipMalloc((void **) &allowed, n);
  hipMemcpy(allowed, &temp, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_board, board[0], sizeof(int) * n * n, hipMemcpyHostToDevice);

  if (col >= n * n)
  {
    total++;
    return 1;
  }

  int nextState = 0;
  for(int k = 0; k < n; k++)
  {
    isAllowedGpu<<<numBlocks, threadsPerBlock>>>(d_board, k, col, n * n, allowed);
    hipMemcpy(&temp, allowed, sizeof(int), hipMemcpyDeviceToHost);
    if(temp == 1)
    {
      board[k][col] = 1;
      nextState = Solver(board, col + 1, n) || nextState;
      board[k][col] = 0;
    }
  }

  return nextState;
}

double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

int main(int argc, char **argv) {
  //  CPU VERSION
  
  const int n = atoi(argv[1]);
  int **board;
  board = (int **) malloc(n * sizeof(int *));
  for (int i = 0; i < n; i++) {
    board[i] = (int *) malloc(n * sizeof(int));
  }
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      board[i][j] = 0;
    }
  }
	// vector<vector<int> > board;
	// board.resize(n, std::vector<int>(n, 0));
	
	srand(1);
	gettimeofday(&startTime, &Idunno);

  if(Solver(board,0,n) == 0)
  {
    printf("No Solution\n");
  	report_running_time();
    return 0;
  }
  printf("\nTotal Solutions(CPU): %d boards\n\n",total);
	report_running_time();

  return 0;

}
